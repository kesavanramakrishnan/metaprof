
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>

__global__ void gemm_kernel(const float* __restrict__ A,
                            const float* __restrict__ B,
                            float* __restrict__ C,
                            int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < N && col < N) {
        float acc = 0.0f;
        for (int k = 0; k < N; ++k) {
            acc += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = acc;
    }
}

int main(int argc, char** argv) {
    int N = 512;
    if (argc > 1) {
        N = std::atoi(argv[1]);
    }
    size_t bytes = static_cast<size_t>(N) * N * sizeof(float);

    float *hA = (float*)malloc(bytes);
    float *hB = (float*)malloc(bytes);
    float *hC = (float*)malloc(bytes);
    for (int i = 0; i < N * N; ++i) {
        hA[i] = 1.0f;
        hB[i] = 1.0f;
    }

    float *dA, *dB, *dC;
    hipMalloc(&dA, bytes);
    hipMalloc(&dB, bytes);
    hipMalloc(&dC, bytes);
    hipMemcpy(dA, hA, bytes, hipMemcpyHostToDevice);
    hipMemcpy(dB, hB, bytes, hipMemcpyHostToDevice);

    dim3 block(16, 16);
    dim3 grid((N + block.x - 1) / block.x, (N + block.y - 1) / block.y);
    gemm_kernel<<<grid, block>>>(dA, dB, dC, N);
    hipDeviceSynchronize();

    hipMemcpy(hC, dC, bytes, hipMemcpyDeviceToHost);
    // print a checksum so the compiler doesn't optimize away everything
    double checksum = 0.0;
    for (int i = 0; i < N * N; ++i) checksum += hC[i];
    std::printf("checksum: %.0f\n", checksum);

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    free(hA);
    free(hB);
    free(hC);
    return 0;
}


